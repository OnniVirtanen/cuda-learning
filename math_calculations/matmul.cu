#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <assert.h>
#include <hip/hip_runtime.h>


#define N 2
#define DEBUG

__global__ void matmul(float* out, float* arr_1, float* arr_2, int n) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    if (row < n && col < n) {
        float value = 0;
        for (int k = 0; k < n; ++k) {
            value += arr_1[row * n + k] * arr_2[k * n + col];
        }
        out[row * n + col] = value;
    }
}

bool is_matrix_equal(float arr_1[N][N], float arr_2[N][N]) {
    for (int i = 0; i < N; ++i) {
        for (int j = 0; j < N; ++j) {
            #ifdef DEBUG 
                printf("arr1[%d][%d]: %f, arr2[%d][%d]: %f \n", i, j, arr_1[i][j], i, j, arr_2[i][j]);
            #endif 
            if (arr_1[i][j] != arr_2[i][j]) {
                return false;
            }
        }
    }
    return true;
}

int main() {
    // Host variables
    float array_1[N][N] = {{1, 2}, {3, 4}};
    float array_2[N][N] = {{5, 6}, {7, 8}};
    float out[N][N] = {0};

    // Device variables
    float* d_array_1;
    float* d_array_2;
    float* d_out;

    // Allocate device memory
    hipMalloc((void**)&d_array_1, N * N * sizeof(float));
    hipMalloc((void**)&d_array_2, N * N * sizeof(float));
    hipMalloc((void**)&d_out, N * N * sizeof(float));

    // Transfer input data from host to device memory
    hipMemcpy(d_array_1, array_1, N * N * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_array_2, array_2, N * N * sizeof(float), hipMemcpyHostToDevice);

    // Executing kernel
    dim3 threadsPerBlock(N, N);
    dim3 numBlocks(1, 1);
    matmul<<<numBlocks, threadsPerBlock>>>(d_out, d_array_1, d_array_2, N);

    // Transfer output data from device to host memory
    hipMemcpy(out, d_out, N * N * sizeof(float), hipMemcpyDeviceToHost);

    // Verification
    float array_expected[N][N] = {{19, 22}, {43, 50}};
    assert(is_matrix_equal(out, array_expected) == true);

    // Deallocate device memory
    hipFree(d_array_1);
    hipFree(d_array_2);
    hipFree(d_out);

    return 0;
}
